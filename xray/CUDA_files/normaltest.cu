#include "hip/hip_runtime.h"
#pragma once

#include <optix.h>
#include <optix_cuda.h>
#include <optixu/optixu_math_namespace.h>
#include "core.cuh";
#include "basematerial.cuh"

using namespace optix;

__device__ float3 evalBSDFLocal(const float3& incoming, const float3& outgoing) {
  return make_float3(0);
}

__device__ float evalPDFLocal(const float3& incoming, const float3& outgoing) {
  return 0.0f;
}

__device__ void sampleLocal(
  hiprandState* rng,
  const float3& incoming,
  float3* outgoingOut,
  float3* bsdfOut,
  float* pdfOut
) {
  *outgoingOut = make_float3(0);
  *bsdfOut = make_float3(0);
  *pdfOut = 0.0f;
}

__device__ void scatter(NormalRayData& rayData, float3 normal, float3 pos) {
  normal.x = fabsf(normal.x);
  normal.y = fabsf(normal.y);
  normal.z = fabsf(normal.z);
	rayData.radiance += normal * rayData.beta;
  rayData.beta = make_float3(0);
}

__device__ __inline__ bool shouldDirectIlluminate() {
  return false;
}