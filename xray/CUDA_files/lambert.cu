#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_cuda.h>
#include <optix_world.h>
#include "core.cuh"
#include "math.cuh"

rtDeclareVariable(float3, albedo, , );

RT_CALLABLE_PROGRAM float3 evalBSDFLocal(const float3& incoming, const float3& outgoing) {
  if (!math::localSameHemisphere(incoming, outgoing)) {
    return make_float3(0);
  }

  return albedo * XRAY_INV_PI;
}

RT_CALLABLE_PROGRAM float evalPDFLocal(const float3& incoming, const float3& outgoing) {
  if (!math::localSameHemisphere(incoming, outgoing)) {
    return 0.0f;
  }

  return math::cosineSampleHemispherePDF(outgoing);
}

RT_CALLABLE_PROGRAM void sampleLocal(
  hiprandState* rng,
  const float3& incoming,
  float3* outgoingOut,
  float3* bsdfOut,
  float* pdfOut
) {
  float3 outgoing = math::cosineSampleHemisphere(rng, incoming.z < 0.0f);

  *outgoingOut = outgoing;
  *bsdfOut = evalBSDFLocal(incoming, outgoing);
  *pdfOut = math::cosineSampleHemispherePDF(outgoing);
}
