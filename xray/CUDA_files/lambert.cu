#include "hip/hip_runtime.h"
#include "bsdf.cuh"

__device__ float3 evalBSDFLocal(const float3& incoming, const float3& outgoing) {
  if (!math::localSameHemisphere(incoming, outgoing)) {
    return make_float3(0);
  }

  return albedo * XRAY_INV_PI;
}

__device__ float evalPDFLocal(const float3& incoming, const float3& outgoing) {
  if (!math::localSameHemisphere(incoming, outgoing)) {
    return 0.0f;
  }

  return math::cosineSampleHemispherePDF(outgoing);
}

__device__ void sampleLocal(
  hiprandState* rng,
  const float3& incoming,
  float3* outgoingOut,
  float3* bsdfOut,
  float* pdfOut
) {
  float3 outgoing = math::cosineSampleHemisphere(rng, incoming.z < 0.0f);

  *outgoingOut = outgoing;
  *bsdfOut = evalBSDFLocal(incoming, outgoing);
  *pdfOut = math::cosineSampleHemispherePDF(outgoing);
}

__device__ __inline__ bool shouldDirectIlluminate() {
  return true;
}